#include "hip/hip_runtime.h"
#define RENDER_GPU

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#endif

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>

#include <hip/hip_vector_types.h>
#include <hip/hip_vector_types.h>
#include <helper_timer.h>

#include "scene/scene.cuh"

#include "utils/render_gpu.cuh"
#include "utils/render_cpu.cuh"

// OpenGL Graphics includes
#ifndef OPENGL_HEADERS
#define OPENGL_HEADERS
#include <helper_gl.h>
#if defined(__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#ifndef glutCloseFunc
#define glutCloseFunc glutWMCloseFunc
#endif
#else
#include <GL/freeglut.h>
#endif
#endif

#ifdef RENDER_GPU
#define WIDTH 1300
#define HEIGHT 800
#else
#define WIDTH 200
#define HEIGHT 100
#endif

GLubyte *h_bitmap;
GLubyte *d_bitmap;

int tx = 16;
int ty = 16;

int nx = WIDTH;
int ny = HEIGHT;

int num_pixels = nx * ny;
size_t bitmap_size = num_pixels * sizeof(GLubyte);

int SPHERES_COUNT = 1000;
int LIGHTS_COUNT = 200;

s_scene d_scene;
s_scene h_scene;

int resolution_horizontal = WIDTH;
int resolution_vertical = HEIGHT;
float fov;

StopWatchInterface *fps_timer = NULL;

float angle_x = 0;
float angle_y = 0;
int state = 1;

clock_t start, stop;
clock_t second_start, second_stop;
double raycasting_time = 0;
double cpu_to_gpu_copying_time = 0;
double gpu_to_cpu_copying_time = 0;

int start_x = -1;
int start_y = -1;

float start_angle_x = 0;
float start_angle_y = 0;

float prev_angle_x = 0;
float prev_angle_y = 0;

int shift_pressed = 0;

int frames;
int fpsCount = 0;
float avgFPS = 1.0f;
int fpsLimit = 1;
double first_second = 2;

char fps[512];

float rand_float(float min, float max)
{
    float random = ((float)rand()) / (float)RAND_MAX;
    float diff = max - min;
    float r = random * diff;
    return min + r;
}

void display()
{
    glClear(GL_COLOR_BUFFER_BIT);
    glLoadIdentity();
    glDrawPixels(WIDTH, HEIGHT, GL_RGB, GL_UNSIGNED_BYTE, h_bitmap);
    glutSwapBuffers();
}

void assign_position(s_positions &positions, int i)
{
    float x = rand_float(-150, 150);
    float y = rand_float(-150, 150);

    float dist = sqrt(x * x + y * y);

    float s_x = x / dist;
    float s_y = y / dist;

    positions.angle[i] = atan2(s_y, s_x);

    positions.x[i] = x;
    positions.y[i] = y;
    positions.z[i] = rand_float(-100, 100);
}

void randomize_scene_variables()
{
    d_scene.camera.origin = make_float3(-1000, 600, 700);

    set_resolution(d_scene.camera, WIDTH, HEIGHT);
    look_at(d_scene.camera, 0, 0, 0);

    h_scene.camera.origin = make_float3(-1000, 600, 700);

    set_resolution(h_scene.camera, WIDTH, HEIGHT);
    look_at(h_scene.camera, 0, 0, 0);

    for (int i = 0; i < SPHERES_COUNT; i++)
    {
        h_scene.spheres.r[i] = rand_float(2, 4);

        assign_position(h_scene.spheres.pos, i);

        h_scene.spheres.color.r[i] = rand_float(0, 1);
        h_scene.spheres.color.g[i] = rand_float(0, 1);
        h_scene.spheres.color.b[i] = rand_float(0, 1);
    }

    for (int i = 0; i < LIGHTS_COUNT; i++)
    {
        assign_position(h_scene.lights.pos, i);
    }
}

void rotate_objects(s_positions *positions, int n, float rotate)
{
    for (int i = 0; i < n; i++)
    {
        float x = positions->x[i];
        float y = positions->y[i];

        float angle = positions->angle[i] + rotate;
        float dist = sqrt(x * x + y * y);

        positions->x[i] = dist * cos(angle);
        positions->y[i] = dist * sin(angle);

        positions->angle[i] = angle;
    }
}

void memcpy_device_to_host()
{
    checkCudaErrors(hipMemcpy(h_bitmap, d_bitmap, bitmap_size, hipMemcpyDeviceToHost));
}

void memcpy_host_to_device()
{
    checkCudaErrors(hipMemcpy(d_scene.spheres.r, h_scene.spheres.r, sizeof(float) * SPHERES_COUNT, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_scene.spheres.pos.x, h_scene.spheres.pos.x, sizeof(float) * SPHERES_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_scene.spheres.pos.y, h_scene.spheres.pos.y, sizeof(float) * SPHERES_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_scene.spheres.pos.z, h_scene.spheres.pos.z, sizeof(float) * SPHERES_COUNT, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_scene.spheres.color.r, h_scene.spheres.color.r, sizeof(float) * SPHERES_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_scene.spheres.color.g, h_scene.spheres.color.g, sizeof(float) * SPHERES_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_scene.spheres.color.b, h_scene.spheres.color.b, sizeof(float) * SPHERES_COUNT, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_scene.lights.pos.x, h_scene.lights.pos.x, sizeof(float) * LIGHTS_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_scene.lights.pos.y, h_scene.lights.pos.y, sizeof(float) * LIGHTS_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_scene.lights.pos.z, h_scene.lights.pos.z, sizeof(float) * LIGHTS_COUNT, hipMemcpyHostToDevice));
}

void malloc_bitmap()
{
    num_pixels = nx * ny;
    bitmap_size = num_pixels * sizeof(GLubyte) * 3;
    free(h_bitmap);
    h_bitmap = new GLubyte[nx * ny * 3];

    hipFree(d_bitmap);
    checkCudaErrors(hipMallocManaged((void **)&d_bitmap, bitmap_size));
}

void malloc_scene()
{
    sdkCreateTimer(&fps_timer);

    h_scene.spheres.pos.x = (float *)malloc(sizeof(float) * SPHERES_COUNT);
    h_scene.spheres.pos.y = (float *)malloc(sizeof(float) * SPHERES_COUNT);
    h_scene.spheres.pos.z = (float *)malloc(sizeof(float) * SPHERES_COUNT);
    h_scene.spheres.pos.angle = (float *)malloc(sizeof(float) * SPHERES_COUNT);

    h_scene.spheres.color.r = (float *)malloc(sizeof(float) * SPHERES_COUNT);
    h_scene.spheres.color.g = (float *)malloc(sizeof(float) * SPHERES_COUNT);
    h_scene.spheres.color.b = (float *)malloc(sizeof(float) * SPHERES_COUNT);

    h_scene.spheres.r = (float *)malloc(sizeof(float) * SPHERES_COUNT);

    h_scene.lights.pos.x = (float *)malloc(sizeof(float) * LIGHTS_COUNT);
    h_scene.lights.pos.y = (float *)malloc(sizeof(float) * LIGHTS_COUNT);
    h_scene.lights.pos.z = (float *)malloc(sizeof(float) * LIGHTS_COUNT);
    h_scene.lights.pos.angle = (float *)malloc(sizeof(float) * LIGHTS_COUNT);

    malloc_bitmap();

    d_scene.spheres.n = SPHERES_COUNT;
    h_scene.spheres.n = SPHERES_COUNT;

    checkCudaErrors(hipMalloc((void **)&d_scene.spheres.pos.x, SPHERES_COUNT * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_scene.spheres.pos.y, SPHERES_COUNT * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_scene.spheres.pos.z, SPHERES_COUNT * sizeof(float)));

    checkCudaErrors(hipMalloc((void **)&d_scene.spheres.color.r, SPHERES_COUNT * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_scene.spheres.color.g, SPHERES_COUNT * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_scene.spheres.color.b, SPHERES_COUNT * sizeof(float)));

    checkCudaErrors(hipMalloc((void **)&d_scene.spheres.r, SPHERES_COUNT * sizeof(float)));

    d_scene.lights.n = LIGHTS_COUNT;
    h_scene.lights.n = LIGHTS_COUNT;

    checkCudaErrors(hipMalloc((void **)&d_scene.lights.pos.x, LIGHTS_COUNT * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_scene.lights.pos.y, LIGHTS_COUNT * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_scene.lights.pos.z, LIGHTS_COUNT * sizeof(float)));
}

void free_scene()
{
    sdkDeleteTimer(&fps_timer);

    free(h_scene.spheres.pos.x);
    free(h_scene.spheres.pos.y);
    free(h_scene.spheres.pos.z);
    free(h_scene.spheres.pos.angle);

    free(h_scene.spheres.color.r);
    free(h_scene.spheres.color.g);
    free(h_scene.spheres.color.b);

    free(h_scene.spheres.r);

    free(h_scene.lights.pos.x);
    free(h_scene.lights.pos.y);
    free(h_scene.lights.pos.z);
    free(h_scene.lights.pos.angle);

    hipFree(d_bitmap);

    hipFree(d_scene.spheres.pos.x);
    hipFree(d_scene.spheres.pos.y);
    hipFree(d_scene.spheres.pos.z);

    hipFree(d_scene.spheres.color.r);
    hipFree(d_scene.spheres.color.g);
    hipFree(d_scene.spheres.color.b);

    hipFree(d_scene.spheres.r);

    hipFree(d_scene.lights.pos.x);
    hipFree(d_scene.lights.pos.y);
    hipFree(d_scene.lights.pos.z);

    free(h_bitmap);
}

void reshape(int w, int h)
{
    glViewport(0, 0, (GLsizei)w, (GLsizei)h);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluOrtho2D(0, w, h, 0);
    glMatrixMode(GL_MODELVIEW);
}

void render_scene()
{
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);

    start = clock();
#ifdef RENDER_GPU
    render_gpu<<<blocks, threads>>>(d_bitmap, nx, ny, d_scene);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
#else
    render_cpu(h_bitmap, nx, ny, h_scene);
#endif
    stop = clock();
    raycasting_time = ((double)(stop - start)) / CLOCKS_PER_SEC;

    start = clock();
#ifdef RENDER_GPU
    memcpy_device_to_host();
#endif
    stop = clock();
    gpu_to_cpu_copying_time = ((double)(stop - start)) / CLOCKS_PER_SEC;
}

void computeFPS()
{
    frames++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&fps_timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&fps_timer);
    }

    second_stop = clock();
    double second_passed = ((double)(second_stop - second_start)) / CLOCKS_PER_SEC + first_second;

    if (second_passed > 1)
    {
        first_second = 0;
        second_start = clock();

#ifdef RENDER_GPU
        sprintf(fps, "Spheres Raycasting: %3.2f fps (Raycasting: %.4f s, CPU->GPU copying: %.6f s, GPU->CPU copying: %.6f s)",
                avgFPS, raycasting_time, cpu_to_gpu_copying_time, gpu_to_cpu_copying_time);
#else
        sprintf(fps, "Spheres Raycasting: %3.2f fps (Raycasting: %.4f s)",
                avgFPS, raycasting_time);
#endif

        std::cout << fps << "\n";
    }

    glutSetWindowTitle(fps);
}

void timer(int)
{
    sdkStartTimer(&fps_timer);
    glutPostRedisplay();

    glutTimerFunc(1000 / 100, timer, 0);

    float angle_diff = -(angle_x - prev_angle_x);

    if (shift_pressed)
    {
        rotate_objects(&(h_scene.lights.pos), LIGHTS_COUNT, angle_diff);
    }
    else
    {
        rotate_objects(&(h_scene.spheres.pos), SPHERES_COUNT, angle_diff);
    }

    prev_angle_x = angle_x;
    prev_angle_y = angle_y;

    render_scene();

#ifdef RENDER_GPU
    start = clock();
    memcpy_host_to_device();
    stop = clock();
    cpu_to_gpu_copying_time = ((double)(stop - start)) / CLOCKS_PER_SEC;
#endif

    sdkStopTimer(&fps_timer);
    computeFPS();
}

// glutMotionFunc() event handler
void mouse(int button, int state, int x, int y)
{
    shift_pressed = glutGetModifiers() & GLUT_ACTIVE_SHIFT;

    if (state == GLUT_DOWN)
    {
        start_x = x;
        start_y = y;

        start_angle_x = angle_x;
        start_angle_y = angle_y;
    }
}

// glutMotionFunc() event handler
void drag(int x, int y)
{
    angle_x = start_angle_x + ((float)(x - start_x) / 300.0);
    angle_y = start_angle_y + ((float)(y - start_y) / 300.0);
}

void setup_opengl(int argc, char *argv[])
{
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GL_DOUBLE);

    glutInitWindowSize(nx, ny);
    glutInitWindowPosition(100, 100);

    int MainWindow = glutCreateWindow("Sphere Raycasting");
    glClearColor(0.0, 0.0, 0.0, 0);

    glutDisplayFunc(display);
    glutMouseFunc(mouse);
    glutReshapeFunc(reshape);
    glutMotionFunc(drag);
    glutTimerFunc(0, timer, 0);
    glutMainLoop();
}

int main(int argc, char *argv[])
{
    std::cerr << "Raycasting a scene of " << SPHERES_COUNT << " spheres and " << LIGHTS_COUNT << " lights with " << nx << "x" << ny;
    std::cerr << " rays in " << tx << "x" << ty << " blocks.\n";

    second_start = clock();

    malloc_scene();
    checkCudaErrors(hipGetLastError());
    randomize_scene_variables();
    memcpy_host_to_device();

    render_scene();

    setup_opengl(argc, argv);

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_scene();

    hipDeviceReset();
}